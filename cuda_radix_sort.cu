#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <fstream>
#include <omp.h>
#include <math.h>

#define DO_CUDA_DEBUG 0 // enables debug prints, disables flipping bits, and uses base 10 for running

#if DO_CUDA_DEBUG
#define ARRAY_PRINT_THRESHOLD 30
#else
#define ARRAY_PRINT_THRESHOLD 20
#endif

#define NUM_THREADS 4 // number of CUDA threads

#if DO_CUDA_DEBUG
#define NUM_BASE 10
#else
#define NUM_BASE 256
#endif
#define COUNT_ARRAY_SIZE NUM_BASE // the count array will always hold the same number of values as the number of digits

/**
 * @brief Prints the array.
 *
 * @param name The name of the array.
 * @param array The array to print.
 * @param arrayLen The length of the array.
 */
static void printArray(const char *name, uint *array, uint arrayLen)
{
    printf("\nArray %s, length %d:", name, arrayLen);
    if (arrayLen <= ARRAY_PRINT_THRESHOLD)
    {
        printf(" [");
        for (uint i = 0; i < arrayLen; i++)
        {
            printf("%d", array[i]);

            if (i != arrayLen - 1)
            {
                printf(", ");
            }
        }
        printf("]\n");
    }
    else
    {
        printf(" Array above printing threshold %d\n", ARRAY_PRINT_THRESHOLD);
    }
}

/**
 * @brief Reads an integer array from a file.
 *
 * @param fileName The name of the file to read from.
 * @param outputNumElements The length of the output array.
 * @note The output array is allocated on the heap and MUST be deallocated by the user.
 *
 * @returns True on failure, false on success.
 */
static uint *readIntArrayFromFile(const char *fileName, uint &outputNumElements)
{
    if (!fileName)
    {
        // bad argument
        return NULL;
    }

    std::vector<uint> output;

    // since this can fail when reading from input
    try
    {
        std::ifstream inputFile(fileName, std::ios_base::in);
        int curInt;
        while (inputFile >> curInt)
        {
            output.push_back((uint)curInt);
        }
    }
    catch (...)
    {
        return NULL; // fail
    }

    if (output.empty())
    {
        // either no integers or a bad file name
        return NULL;
    }

    uint *outputPointer = new uint[output.size()];
    for (uint i = 0; i < (uint)output.size(); i++)
    {
        outputPointer[i] = output[i];
    }
    outputNumElements = output.size();

    return outputPointer; // success
}

/**
 * @brief Flips all the sign bits in an array.
 *
 * @param array The array to flip sign bits in.
 * @param arrayLength The lengh of the arrray.
 */
static void flipSignBits(int *array, uint arrayLength)
{
    // create mask for flipping the sign bit
    uint mask = -1;
    mask -= 1;
    mask = (mask >> 1); // shift one to the right so we flip only the sign bit
    mask = ~mask;

    // flip all the sign bits
    for (uint i = 0; i < arrayLength; i++)
    {
        array[i] ^= mask;
    }
}

/**
 * @brief Finds the maxmimum value in an array at a digit and outputs it.
 *
 * @param array The array to search.
 * @param arrayLen The length of the array.
 * @param digit The digit to find the maximum value for.
 * @param output The output maximum value.
 *
 * @returns `true` on failure, `false` on success.
 */
static bool getMax(const uint *array, const uint arrayLen, uint *output)
{
    if (!array || !output)
    {
        return true; // fail
    }

    uint maxValue = array[0];
    // find the maximum
    for (uint i = 1; i < arrayLen; i++)
    {
        if (maxValue < array[i])
        {
            maxValue = array[i];
        }
    }

    *output = maxValue;
    return false; // success
}

/**
 * @brief Gets the number of digits that this value has.
 *
 * @param value The value to find the number of digits of.
 * @return The number of digits.
 */
static uint getNumDigits(uint value)
{
    uint numDigits = 1; // we should start at one
    value /= NUM_BASE;
    while (value > 0)
    {
        numDigits++;
        value /= NUM_BASE;
    }

    return numDigits;
}

/**
 * @brief Finds the power of a value.
 *
 * @param value The value to find power of.
 * @param exponent The exponent.
 * @return The result of value to the power of exponent.
 */
static unsigned long long myPowUint(uint value, uint exponent)
{
    unsigned long long sum = 1;
#pragma omp parallel for reduction(* : sum)
    for (uint i = 0; i < exponent; i++)
    {
        sum *= value;
    }

    return sum;
}

/**
 * @brief Calculates elapsed time from two times.
 *
 * @param initialTime The initial time.
 * @param finalTime The final time.
 * @returns The elapsed time.
 */
static double calculateElapsedTime(timespec &initialTime, timespec &finalTime)
{
    double output = (finalTime.tv_sec - initialTime.tv_sec) + (finalTime.tv_nsec - initialTime.tv_nsec);
    if (output < 0)
    {
        return 0; // bad time
    }

    return output;
}

/**
 * @brief Checks if the array is sorted.
 *
 * @param array The array to check.
 * @param arrayLen The array length.
 *
 * @returns `true` if the array is sorted, `false` otherwise.
 */
static bool isSorted(int *array, uint arrayLen)
{
    bool output = true;
#pragma omp parallel for
    for (uint i = 1; i < arrayLen; i++)
    {
        if (array[i - 1] > array[i])
        {
            output = false;
        }
    }

    return output;
}

/**
 * @brief Updates the count matrix for the local array.
 *
 * @param countMatrix The countmatrix to modify.
 * @param localArray The local array to read from.
 * @param localArraySize The local array size.
 * @param digit The current digit to update the count matrix on.
 */
__global__ void updateCountMatrix(uint *countMatrix, const uint *localArray,
                                  const uint localArraySize, const uint digit)
{
    // Count the occurrences of each digit at the current place value in the local array
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
#if DO_CUDA_DEBUG // DEBUG: debug print
    printf("DEBUG: index=%d\n", index);
#endif
    if (index < localArraySize)
    {
        uint digitValue = (localArray[index] / digit) % NUM_BASE;
        countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + digitValue]++;

#if DO_CUDA_DEBUG // DEBUG: debug print
        printf("DEBUG: index=%d, blockIdx.x=%d,blockDim.x=%d, threadIdx.x=%d, countmatrix[%d]=%u, digitValue=%u\n",
               index, blockIdx.x,
               blockDim.x, threadIdx.x, (blockIdx.x * COUNT_ARRAY_SIZE) + digitValue,
               countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + digitValue], digitValue);
#endif
    }
}

/**
 * @brief Finds the offset table from the block sums.
 *
 * @param deviceGlobalOffset The device global offset array.
 * @param countMatrix The count matrix.
 */
__global__ void sumOffsetTable(uint *deviceGlobalOffset, uint *countMatrix)
{
    int sum = 0;

    // sum this count array
    for (int i = 0; i < COUNT_ARRAY_SIZE; i++)
    {
        sum += countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + i];
#if DO_CUDA_DEBUG // DEBUG: debug print
        printf("DEBUG: sum=%d, i=%d, blockSumsIdx=%d, blockIdx=%d deviceGlobalOffset[blockIdx.x]=%d\n",
               sum, i, blockIdx.x, blockIdx, deviceGlobalOffset[blockIdx.x]);
#endif
    }
    deviceGlobalOffset[blockIdx.x] = sum;
#if DO_CUDA_DEBUG // DEBUG: debug print
    printf("DEBUG: blockIdx.x=%d, deviceGlobalOffset[blockIdx.x]=%d\n", blockIdx.x, deviceGlobalOffset[blockIdx.x]);
#endif
}

__global__ void shiftOffsetTable(uint *deviceGlobalOffset, uint *newBlockSums, int numBlocks, int iterationNum)
{
#if DO_CUDA_DEBUG // DEBUG: debug print
    printf("DEBUG: shiftOffsetTable iteration=%d\n", iterationNum);
#endif
    if (iterationNum == 0)
    {
        newBlockSums[0] = 0;
        int index = blockIdx.x + 1;
        if (index < numBlocks && blockIdx.x < numBlocks)
        {
            newBlockSums[index] = deviceGlobalOffset[blockIdx.x];
        }
    }
    else
    {
        int powResult = (int)pow(2, iterationNum - 1);
        if (blockIdx.x >= powResult)
        {
            int first = blockIdx.x - powResult;
            int second = blockIdx.x;
#if DO_CUDA_DEBUG // DEBUG: debug print
            printf("DEBUG: blockIdx.x=%d first=%d, second=%d, iteration=%d, pow(2, iterationNum - 1)=%d\n",
                   blockIdx.x, first, second, iterationNum, powResult);
#endif

            newBlockSums[blockIdx.x] = deviceGlobalOffset[first] + deviceGlobalOffset[second];
        }
    }
}

/**
 * @brief Performs the parallel scan algorithm to find prefix sum.
 *
 * @param numBlocks The number of blocks.
 * @param deviceGlobalOffset The global offset.
 */
static void parallelScan(int numBlocks, uint *deviceGlobalOffset)
{
    // run until log2(numBlocks)
    for (int i = 0; i <= (int)log2(numBlocks); i++)
    {
#if DO_CUDA_DEBUG
        printf("DEBUG: CPU LOOP i=%d\n", i);
#endif
        uint *deviceNewBlockSums = NULL;
        hipMallocManaged(&deviceNewBlockSums, sizeof(uint) * numBlocks);
        hipMemset(deviceNewBlockSums, 0, sizeof(uint) * numBlocks);
        shiftOffsetTable<<<numBlocks, 1>>>(deviceGlobalOffset, deviceNewBlockSums, numBlocks, i);
        hipDeviceSynchronize();
        // copy over the new sums to the block sums
        hipMemcpy(deviceGlobalOffset, deviceNewBlockSums, sizeof(uint) * numBlocks,
                   hipMemcpyKind::hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();
        hipFree(deviceNewBlockSums);
        uint *tempCopy = (uint *)malloc(sizeof(uint) * numBlocks);
#if DO_CUDA_DEBUG // DEBUG: debug print
        cudaMemcpy(tempCopy, deviceGlobalOffset, sizeof(uint) * numBlocks,
                   cudaMemcpyKind::cudaMemcpyDeviceToHost);
        printArray("deviceGlobalOffset", tempCopy, numBlocks);
#endif
    }
}

/**
 * @brief Reorders elements into sorted positions.
 *
 * @param countMatrix The count matrix.
 * @param inputArray The input array to read from.
 * @param outputArray The output array to write to.
 * @param deviceGlobalOffset The device global offset array.
 * @param localArraySize The local array size.
 * @param digit The current digit.
 */
__global__ void reorderElements(uint *countMatrix, const uint *inputArray, uint *outputArray,
                                const uint *deviceGlobalOffset, const uint localArraySize,
                                const uint digit)
{
    // Count the occurrences of each digit at the current place value in the local array
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
#if DO_CUDA_DEBUG // DEBUG: debug print
    printf("DEBUG: index=%d\n", index);
#endif
    if (index < localArraySize)
    {
        uint digitValue = (inputArray[index] / digit) % NUM_BASE;
        int localOffset = countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + digitValue];
        int globalOffset = deviceGlobalOffset[blockIdx.x];

        int globalIdx = index - localOffset + globalOffset;

        outputArray[globalIdx] = inputArray[index];

#if DO_CUDA_DEBUG // DEBUG: debug print
        printf("DEBUG: index=%d, blockIdx.x=%d,blockDim.x=%d, threadIdx.x=%d, countmatrix[%d]=%u, digitValue=%u\n",
               index, blockIdx.x,
               blockDim.x, threadIdx.x, (blockIdx.x * COUNT_ARRAY_SIZE) + digitValue,
               countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + digitValue], digitValue);
#endif
    }
}

int main(int argc, char *argv[])
{
    /* shared variables */
    uint inputArraySize = 0, maxDigit = 0;
    unsigned long long maxPossibleValue = 0;
    uint *inputArray = NULL;
    uint *outputArray = NULL;

    // temp arrays for saving results
    int *numValues = NULL;
    int *tempDisplacements = NULL;

    /* radix sort setup */

    if (argc != 2)
    {
        printf("Usage: %s <input_file_name>\n", argv[0]);
        return 1;
    }

    const char *inputFileName = argv[1];

    inputArray = readIntArrayFromFile(inputFileName, inputArraySize);
    if (!inputArray)
    {
        fprintf(stderr, "Could not read array from file %s. File may be empty or not exist\n", inputFileName);
        return 1;
    }

    // initial info
    printf("\nInput file = \"%s\"\n", inputFileName);

    // print the array
    printArray("Initial", inputArray, inputArraySize);

    // flip bits, then do the rest of the setup
#ifndef DO_CUDA_DEBUG // TODO: bring back
    flipSignBits((int *)inputArray, inputArraySize);
#endif

    outputArray = new uint[inputArraySize];
    (void)memset(outputArray, 0, sizeof(uint) * inputArraySize);

    uint maxValue;
    if (getMax(inputArray, inputArraySize, &maxValue))
    {
        fprintf(stderr, "Could not get the maximum value in the input array\n");
        return 1;
    }

    // find out the number of digits in this maximum value
    maxDigit = getNumDigits(maxValue);

    maxPossibleValue = myPowUint(NUM_BASE, maxDigit);

    // the number of blocks we need to handle each array
    uint numBlocks = ceil((double)inputArraySize / NUM_THREADS);
#if DO_CUDA_DEBUG
    printf("DEBUG: numBlocks=%d\n", numBlocks);
#endif

    // the base size of our local array
    uint localArraySize = numBlocks * NUM_THREADS;

    // Create a local count array to store the count of each digit for the current process
    uint *localCountArray = new uint[COUNT_ARRAY_SIZE];
    // Create a local offset array to store the offsets for the current process
    uint *localOffsetArray = new uint[localArraySize];
    // local array to sort
    uint *localArray = new uint[localArraySize];

    hipError_t err = hipError_t::hipSuccess;
    // copy the entire array to the GPU
    uint *deviceInputArray = NULL;
    err = hipMallocManaged(&deviceInputArray, sizeof(uint) * inputArraySize);
    if (err)
    {
        fprintf(stderr, "Could not malloc device input array");
        return 1;
    }
    err = hipMemcpy(deviceInputArray, inputArray, sizeof(uint) * inputArraySize, hipMemcpyKind::hipMemcpyHostToDevice);
    if (err)
    {
        fprintf(stderr, "Could not copy input array to GPU");
        return 1;
    }

    uint *deviceOutputArray = NULL;
    err = hipMallocManaged(&deviceOutputArray, sizeof(uint) * inputArraySize);
    if (err)
    {
        fprintf(stderr, "Could not malloc device output array");
        return 1;
    }
    err = hipMemset(deviceOutputArray, 0, sizeof(uint) * inputArraySize);
    if (err)
    {
        fprintf(stderr, "Could not memset output array");
        return 1;
    }

    // Allocate deviceCountMatrix and offsetMatrix as contiguous blocks
    uint *deviceCountMatrix = NULL;
    err = hipMallocManaged(&deviceCountMatrix, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not malloc count matrix on GPU");
        return 1;
    }
    err = hipMemset(deviceCountMatrix, 0, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not memset count matrix");
        return 1;
    }

    // initialize offset matrix
    uint *offsetMatrix = NULL;
    err = hipMallocManaged(&offsetMatrix, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not malloc offset matrix on GPU");
        return 1;
    }
    err = hipMemset(offsetMatrix, 0, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not memset offset matrix");
        return 1;
    }

    timespec startTime, finalTime;
    clock_gettime(CLOCK_REALTIME, &startTime);

    for (unsigned long long digit = 1; digit < maxPossibleValue; digit *= NUM_BASE)
    {
        /* GPU */

        // for every block, make sure that it gets values copied from CPU to the GPU
        // NOTE: we do not need to copy the input array value here because we have the whole input array on device

        // perform counts on the array for this digit
        updateCountMatrix<<<numBlocks, NUM_THREADS>>>(deviceCountMatrix,
                                                      deviceInputArray, inputArraySize, digit);
        uint *debugCountMatrix = (uint *)malloc(sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
        hipMemcpy(debugCountMatrix, deviceCountMatrix, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE,
                   hipMemcpyKind::hipMemcpyDeviceToHost);
        printArray("debugCountMatrix", debugCountMatrix, numBlocks * COUNT_ARRAY_SIZE);

        uint *deviceGlobalOffset = NULL;
        hipMallocManaged(&deviceGlobalOffset, sizeof(uint) * numBlocks);
        // perform scan algorithm

        // find initial prefix sums
        sumOffsetTable<<<numBlocks, 1>>>(deviceGlobalOffset, deviceCountMatrix);
        hipDeviceSynchronize();

        // perform parallel scan algorithm
        parallelScan(numBlocks, deviceGlobalOffset);

        // reorder elements back into the original array
        reorderElements<<<numBlocks, NUM_THREADS>>>(deviceCountMatrix, deviceInputArray, deviceOutputArray, deviceGlobalOffset,
                                                    localArraySize, digit);

        // swap the pointers
        uint *temp = deviceInputArray;
        deviceInputArray = deviceOutputArray;
        deviceOutputArray = temp;

        hipFree(deviceGlobalOffset);
    }

    hipDeviceSynchronize(); // DEBUG: remove eventually?
    clock_gettime(CLOCK_REALTIME, &finalTime);
    double elapsedTime = calculateElapsedTime(startTime, finalTime);

    // swap the pointers again :)
    if (maxPossibleValue > 1)
    {
        uint *temp = deviceInputArray;
        deviceInputArray = deviceOutputArray;
        deviceOutputArray = temp;
    }

    // copy back from device
    err = hipMemcpy(outputArray, deviceOutputArray, sizeof(uint) * inputArraySize, hipMemcpyKind::hipMemcpyDeviceToHost);
    if (err)
    {
        fprintf(stderr, "Could not copy device output array to host");
        return 1;
    }

// save time
#ifndef DO_CUDA_DEBUG // TODO: bring back
    flipSignBits((int *)outputArray, inputArraySize);
#endif

    printArray("Final", outputArray, inputArraySize);

    if (isSorted((int *)outputArray, inputArraySize))
    {
        printf("\n\nThe array is sorted in %lf second(s).\n", elapsedTime);
    }
    else
    {
        printf("\n\nThe array is not sorted.\n");
        return 1;
    }

    // delete shared values
    hipFree(deviceInputArray);
    delete[] inputArray;
    inputArray = NULL;
    delete[] outputArray;
    outputArray = NULL;
    delete[] numValues;
    numValues = NULL;
    delete[] tempDisplacements;
    tempDisplacements = NULL;
    delete[] localCountArray;
    localCountArray = NULL;
    delete[] localOffsetArray;
    localOffsetArray = NULL;
    delete[] deviceCountMatrix;
    deviceCountMatrix = NULL;
    delete[] deviceCountMatrix;
    deviceCountMatrix = NULL;
    delete[] offsetMatrix;
    offsetMatrix = NULL;
    delete[] offsetMatrix;
    offsetMatrix = NULL;
    delete[] localArray;
    localArray = NULL;

    return 0;
}
