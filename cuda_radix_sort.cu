#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <fstream>
#include <omp.h>
#include <math.h>

#define ARRAY_PRINT_THRESHOLD 20

// #define NUM_BLOCKS 512
#define NUM_THREADS 4

#define NUM_BASE 10
#define COUNT_ARRAY_SIZE NUM_BASE // the count array will always hold the same number of values as the number of digits
#define INITIAL_ARRAY_SIZE 20

/**
 * @brief Prints the array.
 *
 * @param name The name of the array.
 * @param array The array to print.
 * @param arrayLen The length of the array.
 */
static void printArray(const char *name, uint *array, uint arrayLen)
{
    printf("\nArray %s, length %d:", name, arrayLen);
    if (arrayLen <= ARRAY_PRINT_THRESHOLD)
    {
        printf(" [");
        for (uint i = 0; i < arrayLen; i++)
        {
            printf("%d", array[i]);

            if (i != arrayLen - 1)
            {
                printf(", ");
            }
        }
        printf("]\n");
    }
    else
    {
        printf(" Array above printing threshold %d\n", ARRAY_PRINT_THRESHOLD);
    }
}

/**
 * @brief Reads an integer array from a file.
 *
 * @param fileName The name of the file to read from.
 * @param outputNumElements The length of the output array.
 * @note The output array is allocated on the heap and MUST be deallocated by the user.
 *
 * @returns True on failure, false on success.
 */
static uint *readIntArrayFromFile(const char *fileName, uint &outputNumElements)
{
    if (!fileName)
    {
        // bad argument
        return NULL;
    }

    std::vector<uint> output;

    // since this can fail when reading from input
    try
    {
        std::ifstream inputFile(fileName, std::ios_base::in);
        int curInt;
        while (inputFile >> curInt)
        {
            output.push_back((uint)curInt);
        }
    }
    catch (...)
    {
        return NULL; // fail
    }

    if (output.empty())
    {
        // either no integers or a bad file name
        return NULL;
    }

    uint *outputPointer = new uint[output.size()];
    for (uint i = 0; i < (uint)output.size(); i++)
    {
        outputPointer[i] = output[i];
    }
    outputNumElements = output.size();

    return outputPointer; // success
}

/**
 * @brief Flips all the sign bits in an array.
 *
 * @param array The array to flip sign bits in.
 * @param arrayLength The lengh of the arrray.
 */
static void flipSignBits(int *array, uint arrayLength)
{
    // create mask for flipping the sign bit
    uint mask = -1;
    mask -= 1;
    mask = (mask >> 1); // shift one to the right so we flip only the sign bit
    mask = ~mask;

    // flip all the sign bits
    for (uint i = 0; i < arrayLength; i++)
    {
        array[i] ^= mask;
    }
}

/**
 * @brief Finds the maxmimum value in an array at a digit and outputs it.
 *
 * @param array The array to search.
 * @param arrayLen The length of the array.
 * @param digit The digit to find the maximum value for.
 * @param output The output maximum value.
 *
 * @returns `true` on failure, `false` on success.
 */
static bool getMax(const uint *array, const uint arrayLen, uint *output)
{
    if (!array || !output)
    {
        return true; // fail
    }

    uint maxValue = array[0];
    // find the maximum
    for (uint i = 1; i < arrayLen; i++)
    {
        if (maxValue < array[i])
        {
            maxValue = array[i];
        }
    }

    *output = maxValue;
    return false; // success
}

/**
 * @brief Gets the number of digits that this value has.
 *
 * @param value The value to find the number of digits of.
 * @return The number of digits.
 */
static uint getNumDigits(uint value)
{
    uint numDigits = 1; // we should start at one
    value /= NUM_BASE;
    while (value > 0)
    {
        numDigits++;
        value /= NUM_BASE;
    }

    return numDigits;
}

/**
 * @brief Finds the power of a value.
 *
 * @param value The value to find power of.
 * @param exponent The exponent.
 * @return The result of value to the power of exponent.
 */
static unsigned long long myPowUint(uint value, uint exponent)
{
    unsigned long long sum = 1;
#pragma omp parallel for reduction(* : sum)
    for (uint i = 0; i < exponent; i++)
    {
        sum *= value;
    }

    return sum;
}

/**
 * @brief Calculates elapsed time from two times.
 *
 * @param initialTime The initial time.
 * @param finalTime The final time.
 * @returns The elapsed time.
 */
static double calculateElapsedTime(timespec &initialTime, timespec &finalTime)
{
    double output = (finalTime.tv_sec - initialTime.tv_sec) + (finalTime.tv_nsec - initialTime.tv_nsec);
    if (output < 0)
    {
        return 0; // bad time
    }

    return output;
}

/**
 * @brief Checks if the array is sorted.
 *
 * @param array The array to check.
 * @param arrayLen The array length.
 *
 * @returns `true` if the array is sorted, `false` otherwise.
 */
static bool isSorted(int *array, uint arrayLen)
{
    bool output = true;
#pragma omp parallel for
    for (uint i = 1; i < arrayLen; i++)
    {
        if (array[i - 1] > array[i])
        {
            output = false;
        }
    }

    return output;
}

/**
 * @brief Updates the count matrix for the local array.
 *
 * @param countMatrix The countmatrix to modify.
 * @param countMatrixStart The starting index into the countMatrix to use.
 * @param localArray The local array to read from.
 * @param localArrayStart The starting index into the localArray.
 * @param localArraySize The local array size.
 * @param digit The current digit to update the count matrix on.
 */
__global__ void updateCountMatrix(uint *countMatrix, const uint countMatrixStart, const uint *localArray,
                                  const uint localArrayStart, const uint localArraySize, const uint digit)
{
    // Count the occurrences of each digit at the current place value in the local array
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    printf("DEBUG: index=%d\n", index);

    if (index < localArraySize)
    {
        uint digitValue = (localArray[index] / digit) % NUM_BASE;
        countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + digitValue]++;

        printf("DEBUG: index=%d, blockIdx.x=%d,blockDim.x=%d, threadIdx.x=%d, countmatrix[%d]=%u, digitValue=%u\n",
               index, blockIdx.x,
               blockDim.x, threadIdx.x, (blockIdx.x * COUNT_ARRAY_SIZE) + digitValue,
               countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + digitValue], digitValue);
    }
}

/**
 * @brief Silly inefficient offset calculation.
 *
 * @param blockSums
 * @param countMatrix
 * @return __global__
 */
__global__ void sumOffsetTable(uint *blockSums, uint *countMatrix)
{
    int sum = 0;
    for (int i = 0; i < COUNT_ARRAY_SIZE; i++)
    {
        sum += countMatrix[(blockIdx.x * COUNT_ARRAY_SIZE) + i];
        printf("DEBUG: sum=%d, i=%d, blockSumsIdx=%d, blockIdx=%d blockSums[blockIdx.x]=%d\n",
               sum, i, blockIdx.x, blockIdx, blockSums[blockIdx.x]);
    }
    blockSums[blockIdx.x] = sum;
    printf("DEBUG: blockIdx.x=%d, blockSums[blockIdx.x]=%d\n", blockIdx.x, blockSums[blockIdx.x]);
}

__global__ void shiftOffsetTable(uint *blockSums, uint *newBlockSums, int numBlocks, int iterationNum)
{
    printf("DEBUG: shiftOffsetTable iteration=%d\n", iterationNum);
    if (iterationNum == 0)
    {
        int index = blockIdx.x + 1;
        if (index < numBlocks)
        {
            newBlockSums[index] = blockSums[blockIdx.x];
        }
    }
    else
    {
        int first = blockIdx.x;
        int second = blockIdx.x + pow(2, iterationNum - 1);
        printf("DEBUG: first=%d, second=%d, iteration=%d, myPowInt(2, iterationNum - 1)=%d\n",
               first, second, iterationNum, pow(2, iterationNum - 1));
    }
}

int main(int argc, char *argv[])
{
    /* shared variables */
    uint inputArraySize = 0, maxDigit = 0;
    unsigned long long maxPossibleValue = 0;
    uint *inputArray = NULL;
    uint *outputArray = NULL;

    // temp arrays for saving results
    int *numValues = NULL;
    int *tempDisplacements = NULL;

    /* radix sort setup */

    if (argc != 2)
    {
        printf("Usage: %s <input_file_name>\n", argv[0]);
        return 1;
    }

    const char *inputFileName = argv[1];

    inputArray = readIntArrayFromFile(inputFileName, inputArraySize);
    if (!inputArray)
    {
        fprintf(stderr, "Could not read array from file %s. File may be empty or not exist\n", inputFileName);
        return 1;
    }

    // initial info
    printf("\nInput file = \"%s\"\n", inputFileName);

    // print the array
    printArray("Initial", inputArray, inputArraySize);

    // flip bits, then do the rest of the setup
#if 0 // TODO: bring back
    flipSignBits((int *)inputArray, inputArraySize);
#endif

    outputArray = new uint[inputArraySize];
    (void)memset(outputArray, 0, sizeof(uint) * inputArraySize);

    uint maxValue;
    if (getMax(inputArray, inputArraySize, &maxValue))
    {
        fprintf(stderr, "Could not get the maximum value in the input array\n");
        return 1;
    }

    // find out the number of digits in this maximum value
    maxDigit = getNumDigits(maxValue);

    // FIXME: evil max possible value, don't like this
    maxPossibleValue = myPowUint(NUM_BASE, maxDigit);

    // the number of blocks we need to handle each array
    uint numBlocks = ceil((double)inputArraySize / NUM_THREADS);
    printf("DEBUG: numBlocks=%d\n", numBlocks);

    // the base size of our local array
    uint localArraySize = numBlocks * NUM_THREADS;

    // Create a local count array to store the count of each digit for the current process
    uint *localCountArray = new uint[COUNT_ARRAY_SIZE];
    // Create a local offset array to store the offsets for the current process
    uint *localOffsetArray = new uint[localArraySize];
    // local array to sort
    uint *localArray = new uint[localArraySize];

    hipError_t err = hipError_t::hipSuccess;
    // copy the entire array to the GPU
    uint *deviceInputArray = NULL;
    err = hipMallocManaged(&deviceInputArray, sizeof(uint) * inputArraySize);
    if (err)
    {
        fprintf(stderr, "Could not malloc device input array");
        return 1;
    }
    err = hipMemcpy(deviceInputArray, inputArray, sizeof(uint) * inputArraySize, hipMemcpyKind::hipMemcpyHostToDevice);
    if (err)
    {
        fprintf(stderr, "Could not copy input array to GPU");
        return 1;
    }

    // Allocate deviceCountMatrix and offsetMatrix as contiguous blocks
    uint *deviceCountMatrix = NULL;
    err = hipMallocManaged(&deviceCountMatrix, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not malloc count matrix on GPU");
        return 1;
    }
    err = hipMemset(deviceCountMatrix, 0, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not memset count matrix");
        return 1;
    }

    // initialize offset matrix
    uint *offsetMatrix = NULL;
    err = hipMallocManaged(&offsetMatrix, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not malloc offset matrix on GPU");
        return 1;
    }
    err = hipMemset(offsetMatrix, 0, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not memset offset matrix");
        return 1;
    }

#if 0
    // initialize local array matrix
    uint *deviceLocalArrayMatrix = NULL;
    err = hipMallocManaged(&deviceLocalArrayMatrix, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not malloc local array matrix on GPU");
        return 1;
    }
    err = hipMemset(deviceLocalArrayMatrix, 0, sizeof(uint) * numBlocks * COUNT_ARRAY_SIZE);
    if (err)
    {
        fprintf(stderr, "Could not memset local array matrix");
        return 1;
    }
#endif

    timespec startTime, finalTime;
    clock_gettime(CLOCK_REALTIME, &startTime);

#if 1

    for (unsigned long long digit = 1; digit < maxPossibleValue; digit *= NUM_BASE)
    {
        /* GPU */

        // for every block, make sure that it gets values copied from CPU to the GPU
        // NOTE: we do not need to copy the input array value here because we have the whole input array on device

        // perform counts on the array for this digit
        updateCountMatrix<<<numBlocks, NUM_THREADS>>>(deviceCountMatrix, 0,
                                                      deviceInputArray, 0, inputArraySize, digit);

        uint *deviceBlockSums = NULL;
        hipMallocManaged(&deviceBlockSums, sizeof(uint) * numBlocks);
        // perform scan algorithm

        // find initial prefix sums
        // NOTE: kinda innefficient, but this is more parallel
        sumOffsetTable<<<numBlocks, 1>>>(deviceBlockSums, deviceCountMatrix);
        hipDeviceSynchronize();

        for (int i = 0; i <= numBlocks / 2; i++)
        {
            printf("DEBUG: CPU LOOP i=%d\n", i);
            uint *deviceNewBlockSums = NULL;
            hipMallocManaged(&deviceBlockSums, sizeof(uint) * numBlocks);
            shiftOffsetTable<<<numBlocks, 1>>>(deviceBlockSums, deviceNewBlockSums, numBlocks, i);
            hipDeviceSynchronize();
            // copy over the new sums to the block sums
            hipMemcpy(deviceBlockSums, deviceNewBlockSums, sizeof(uint) * numBlocks,
                       hipMemcpyKind::hipMemcpyDeviceToDevice);
            hipDeviceSynchronize();
        }

        // TODO: housekeeping?? whatever equivalent to what MPI does here

        // TODO: compute offsets

        // TODO: compute local offsets

        // TODO: gather values/place them from in the GPU

#if 0
        // scatter the input array into local arrays
        MPI_Scatterv(inputArray, numValues, tempDisplacements, MPI_UNSIGNED,
                     localArray, (int)localArraySize, MPI_UNSIGNED, 0, comm);

        // update the local count array as the matrix
        updateCountMatrix(localCountArray, localArray, localArraySize, digit);

        // Gather localCountArray into countMatrix
        MPI_Gather(localCountArray, COUNT_ARRAY_SIZE, MPI_UNSIGNED,
                   deviceCountMatrix, COUNT_ARRAY_SIZE, MPI_UNSIGNED, 0, comm);

        MPI_Bcast(deviceCountMatrix, nproc * COUNT_ARRAY_SIZE, MPI_UNSIGNED, 0, comm);

        // compute offsets
        computeOffsets(deviceCountMatrix, nproc, tempDisplacements, COUNT_ARRAY_SIZE, offsetMatrix);

        // compute local offsets
        computeLocalOffsets(localArray, localArraySize, offsetMatrix,
                            COUNT_ARRAY_SIZE, rank, localOffsetArray, digit);

        uint *tempOffsetArray = new uint[inputArraySize];
        MPI_Gatherv(localOffsetArray, localArraySize, MPI_UNSIGNED, tempOffsetArray,
                    numValues, tempDisplacements, MPI_UNSIGNED, 0, comm);

        hipDeviceSynchronize();

        /* CPU */
        // do the move values
        placeValuesFromOffset(inputArray, inputArraySize, tempOffsetArray, outputArray);

        // Swap inputArray and outputArray pointers
        uint *temp = inputArray;
        inputArray = outputArray;
        outputArray = temp;

        delete[] tempOffsetArray;
        tempOffsetArray = NULL;
#endif

        hipFree(deviceBlockSums);
    }
#endif

    hipDeviceSynchronize(); // DEBUG: remove eventually?
    clock_gettime(CLOCK_REALTIME, &finalTime);
    double elapsedTime = calculateElapsedTime(startTime, finalTime);

    // swap the pointers again :)
    if (maxPossibleValue > 1)
    {
        uint *temp = inputArray;
        inputArray = outputArray;
        outputArray = temp;
    }

// save time
#if 0 // TODO: bring back
    flipSignBits((int *)outputArray, inputArraySize);
#endif

    printArray("Final", outputArray, inputArraySize);

    if (isSorted((int *)outputArray, inputArraySize))
    {
        printf("\n\nThe array is sorted in %lf second(s).\n", elapsedTime);
    }
    else
    {
        printf("\n\nThe array is not sorted.\n");
        return 1;
    }

    // delete shared values
    hipFree(deviceInputArray);
    delete[] inputArray;
    inputArray = NULL;
    delete[] outputArray;
    outputArray = NULL;
    delete[] numValues;
    numValues = NULL;
    delete[] tempDisplacements;
    tempDisplacements = NULL;
    delete[] localCountArray;
    localCountArray = NULL;
    delete[] localOffsetArray;
    localOffsetArray = NULL;
    delete[] deviceCountMatrix;
    deviceCountMatrix = NULL;
    delete[] deviceCountMatrix;
    deviceCountMatrix = NULL;
    delete[] offsetMatrix;
    offsetMatrix = NULL;
    delete[] offsetMatrix;
    offsetMatrix = NULL;
    delete[] localArray;
    localArray = NULL;

    return 0;
}

#if 0

int main(int argc, char *argv[])
{
    // call multiply array here
    // question only
    int *deviceArray = NULL;
    int *hostArray = (int *)malloc(sizeof(int) * ARRAY_LEN);
    if (!hostArray)
    {
        fprintf(stderr, "Could not allocate host array\n");
        return 1;
    }

    // initialize array
    for (int i = 0; i < ARRAY_LEN; i++)
    {
        hostArray[i] = i;
    }

    hipMallocManaged(&deviceArray, sizeof(int) * ARRAY_LEN);
    cudaCheckErrors("Failed to initialize device array");

    // copy array to device
    hipMemcpy(deviceArray, hostArray, sizeof(int) * ARRAY_LEN, hipMemcpyKind::hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy from host to device");

    // perform multiply
    multiplyArrayBy<<<512, 1024>>>(deviceArray, ARRAY_LEN, 10);
    hipDeviceSynchronize();
    cudaCheckErrors("Kernel function multiplyArrayBy failed");

    // copy array back to host
    hipMemcpy(hostArray, deviceArray, sizeof(int) * ARRAY_LEN, hipMemcpyKind::hipMemcpyDeviceToHost);
    cudaCheckErrors("Failed to copy array from device to host");

    printArray("Final", hostArray, ARRAY_LEN);

    free(hostArray);
    hipFree(deviceArray);

    return 0;
} /* main */

#endif